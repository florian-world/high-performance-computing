#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <vector>

/// Memory-bound dummy kernel. Do not edit.
__global__ void fastKernel(const double *a, double *b, int M) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= M)
        return;

    b[idx] = 10.0 * a[idx];
}

/// Compute-bound dummy kernel. Do not edit.
__global__ void slowKernel(const double *a, double *b, int M) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= M)
        return;

    double x = a[idx];
    for (int i = 0; i < 10000; ++i)
        x *= 1.01;

    b[idx] = (x != 0.1231 ? 10.0 : -1.0) * a[idx];
}

/// Check whether `bHost` contains the correct result. Do not edit.
void checkResults(const double *bHost, int N) {
    for (int i = 0; i < N; ++i) {
        if (bHost[i] != 100.0 * i) {
            printf("Incorrect value for i=%d:  value before kernel=%.1f  "
                   "expected after=%.1f  now=%.1f\n",
                   i, 10.0 * i, 100. * i, bHost[i]);
            exit(1);
        }
    }
}




/// Asynchronously, and in chunks, copy the array to the device, execute the
/// kernel and copy the result back.
template <typename Kernel>
void runAsync(const char *kernelName, Kernel kernel, int N, int chunkSize, int numStreams) {
    double *aHost, *aDev;
    double *bHost, *bDev;

    CUDA_CHECK(hipHostMalloc(&aHost, N * sizeof(double)));
    CUDA_CHECK(hipHostMalloc(&bHost, N * sizeof(double)));
    for (int i = 0; i < N; ++i)
        aHost[i] = 10.0 * i;

    hipStream_t* streams = new hipStream_t[numStreams];

    for (int i = 0; i < numStreams; ++i)
        hipStreamCreate(streams + i);

    // TODO 3.a) Allocate chunks and create streams.

    // hipStream_t stream; // Declaring the stream variable
    // hipStreamCreate(&stream); // Creating the stream
    // // Assigning Stream to kernel launch
    // myKernel<<grid, shmem, stream>>(args);
    // // Checking if the stream has finished
    // if (hipStreamQuery(stream) == hipSuccess) cout << "Finished";
    // // Waiting for finalization
    // hipStreamSynchronize(stream);
    // // De-allocating memory
    // hipStreamDestroy(stream);

    CUDA_CHECK(hipMalloc(&aDev, N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&bDev, N * sizeof(double)));


    // Instead of benchmark() we use a simplified measure() which invokes the
    // function only once (to get a cleaner profiling information).
    double dt = measure([&]() {
        // TODO 3.b) (1) Upload `a`, (2) launch the kernel, and (3) download
        //           `b` in chunks of size `chunkSize`.
        //           Use streams in a cyclic fashion.
        //
        //           Note: you can use CUDA_CHECK and CUDA_LAUNCH_EX from
        //           utils.h for error checking.


        // error @  12500000
        //     N = 100000000
        // chunk = 100000000
        
        int curStream = 0;

        printf("Launching with chunkSize %d, numStreams = %d, N = %d\n", chunkSize, numStreams, N);
        for (int j = 0; j < N; j += chunkSize) {
            int curSize = std::min(chunkSize, N - j);

            printf("Treating %d now\n", curSize);

            CUDA_CHECK(hipMemcpyAsync(aDev + j, aHost + j, curSize, hipMemcpyHostToDevice, streams[curStream]));
            int threads = 1024;
            int maxBlocks = 65'536;
            int blocks = (curSize + threads - 1) / threads;
            for (int i = 0; i < blocks; i += maxBlocks) {
                int curN = std::min(maxBlocks*threads, N -j - i*threads);
                int curStart = j + i * threads;
                int curBlocks = std::min(maxBlocks, blocks - i);
                printf("Launching kernel in stream %3d to compute from %12d to  %12d with %4d threads and %5d blocks on data of size %12d\n",
                curStream, curStart, curStart + curN - 1, threads, curBlocks, curN);
                CUDA_LAUNCH_EX(kernel, curBlocks, threads, 0, streams[curStream],
                               aDev + curStart, bDev + curStart, curN);
            }
            CUDA_CHECK(hipMemcpyAsync(bHost + j, bDev + j, curSize, hipMemcpyDeviceToHost, streams[curStream]));

            curStream = (curStream + 1) % numStreams;
        }

        for (int i = 0; i < numStreams; ++i)
            hipStreamSynchronize(streams[i]);

        // TODO 3.b) Synchronize the streams.
    });

    checkResults(bHost, N);

    printf("async %s  N=%9d  chunkSize=%9d  numStreams=%d  time=%fs\n",
           kernelName, N, chunkSize, numStreams, dt);

    // TODO: 3.a) Deallocate chunks and destroy streams.

    for (int i = 0; i < numStreams; ++i)
        hipStreamDestroy(streams[i]);
    delete[] streams;

    CUDA_CHECK(hipFree(aDev));
    CUDA_CHECK(hipFree(bDev));
    CUDA_CHECK(hipHostFree(bHost));
    CUDA_CHECK(hipHostFree(aHost));
}





/// Synchronously copy the whole array to the device, execute the kernel and
/// copy the result back. Do not edit.
template <typename Kernel>
void runSync(const char *kernelName, Kernel kernel, int N) {
    double *aHost;
    double *bHost;
    double *aDev;
    double *bDev;

    CUDA_CHECK(hipHostMalloc(&aHost, N * sizeof(double)));
    CUDA_CHECK(hipHostMalloc(&bHost, N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&aDev, N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&bDev, N * sizeof(double)));
    for (int i = 0; i < N; ++i)
        aHost[i] = 10.0 * i;

    // Host -> device.
    double dt1 = measure([&]() {
        CUDA_CHECK(hipMemcpy(aDev, aHost, N * sizeof(double), hipMemcpyHostToDevice));
    });
    // Kernel.
    double dt2 = measure([&]() {
        // We cannot execute more than maxBlocks blocks, so we split the work
        // into multiple launches. That's another reason for using chunks.
        int threads = 1024;
        int maxBlocks = 65'536;
        int blocks = (N + threads - 1) / threads;
        for (int i = 0; i < blocks; i += maxBlocks) {
            CUDA_LAUNCH(kernel, std::min(maxBlocks, blocks - i), threads,
                        aDev + i * threads, bDev + i * threads, std::min(maxBlocks*threads, N - i*threads));
        }
    });
    // Device -> host.
    double dt3 = measure([&]() {
        CUDA_CHECK(hipMemcpy(bHost, bDev, N * sizeof(double), hipMemcpyDeviceToHost));
    });

    checkResults(bHost, N);

    printf("sync  %s  N=%9d  upload=%fs  kernel=%fs  download=%fs  total=%fs\n",
           kernelName, N, dt1, dt2, dt3, dt1 + dt2 + dt3);

    CUDA_CHECK(hipFree(bDev));
    CUDA_CHECK(hipFree(aDev));
    CUDA_CHECK(hipHostFree(bHost));
    CUDA_CHECK(hipHostFree(aHost));
}

/// Selection of runs to use for profiling.
void profile() {
    runSync("fastKernel", fastKernel, 100'000'000);
    runAsync("fastKernel", fastKernel, 100'000'000, 10'000'000, 4);
    runSync("slowKernel", slowKernel, 100'000'000);
    runAsync("slowKernel", slowKernel, 100'000'000, 10'000'000, 4);
    hipProfilerStop();
}

/// Selection of runs to use for benchmarking.
void runBenchmarks() {
    runSync("fastKernel", fastKernel, 1'000'000);
    runSync("fastKernel", fastKernel, 100'000'000);
    runAsync("fastKernel", fastKernel, 100'000'000, 100'000'000, 1);
    runAsync("fastKernel", fastKernel, 100'000'000, 10'000'000, 4);
    runAsync("fastKernel", fastKernel, 100'000'000, 10'000'000, 8);
    runAsync("fastKernel", fastKernel, 100'000'000, 1'000'000, 4);
    runAsync("fastKernel", fastKernel, 100'000'000, 1'000'000, 8);
    printf("\n");

    runSync("slowKernel", slowKernel, 1'000'000);
    runSync("slowKernel", slowKernel, 100'000'000);
    runAsync("slowKernel", slowKernel, 100'000'000, 100'000'000, 1);
    runAsync("slowKernel", slowKernel, 100'000'000, 10'000'000, 4);
    runAsync("slowKernel", slowKernel, 100'000'000, 10'000'000, 8);
    runAsync("slowKernel", slowKernel, 100'000'000, 1'000'000, 4);
    runAsync("slowKernel", slowKernel, 100'000'000, 1'000'000, 8);
}

int main() {
    // TODO: 3.c.) Enable `profile` and disable `runBenchmarks` to get a
    //             cleaner profiling information.

    // profile();
    runBenchmarks();
}
