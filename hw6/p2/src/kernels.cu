#include "hip/hip_runtime.h"
#include "kernels.h"

#include <cstdio>
#include <cassert>

/// Initialize reaction states.
__global__ void initializationKernel(
        short Sa, short Sb, short *x,
        float *t, int numSamples, int numIters)
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numSamples)
        return;

    // Every sample starts with (Sa, Sb) at t=0.
    t[idx] = 0.f;
    x[0 * numIters * numSamples + idx] = Sa;
    x[1 * numIters * numSamples + idx] = Sb;
}


/// Reaction simulation. This kernel uses precomputed random uniform samples
/// (from 0 to 1) to compute up to `numIters` steps of the SSA algorithm. The
/// values of Sa and Sb are stored in `x`, the time values in `t`. Buffer
/// `iters` stores the number of performed iterations, and `isSampleDone`
/// whether or not the sample has reach the final state (t >= endTime).
__global__ void dimerizationKernel(
        int pass, const float *u,
        short *x, float *t, int *iters, char *isSampleDone,
        float endTime, int omega, int numIters, int numSamples,
        double* trajSaThreads, double* trajSbThreads, int* ntrajThreads, int nbins, double bin_dt)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int rngOffset = blockIdx.x * blockDim.x * 2 * numIters + threadIdx.x;

    if (idx >= numSamples)
        return;

    // Reaction rates.
    const float k1 = 1;
    const float k2 = 1;
    const float k3 = 0.2f / omega;
    const float k4 = 20.f * omega;

    // State variables.
    float time;
    float Sa, Sb;

    // Load state.
    const bool continuation = pass > 0 && !isSampleDone[idx];
    if (continuation) {
        Sa   = x[0 * numIters * numSamples + (numIters - 1) * numSamples + idx];
        Sb   = x[1 * numIters * numSamples + (numIters - 1) * numSamples + idx];
        time = t[(numIters - 1) * numSamples + idx];
    } else {
        Sa   = x[0 * numIters * numSamples + idx];
        Sb   = x[1 * numIters * numSamples + idx];
        time = t[idx];
    }

    // Simulation loop.
    int iter;
    for (iter = 0; time < endTime && iter < numIters && (pass == 0 || !isSampleDone[idx]); ++iter) {
        // Accumulated propensities.
        const float a1 = k1*Sa;
        const float a2 = a1 + k2*Sb;
        const float a3 = a2 + k3*Sa*Sb;
        const float a4 = a3 + k4;
        const float a0 = a4;

        time -= 1 / a0 * log(u[rngOffset]);
        rngOffset += blockDim.x;

        const float beta = a0 * u[rngOffset];
        rngOffset += blockDim.x;

        const int d1 = (beta < a1);
        const int d2 = (beta >= a1 && beta < a2);
        const int d3 = (beta >= a2 && beta < a3);
        const int d4 = (beta >= a3);

        Sa += -d1 + d3;
        Sb += -d2 - d3 + d4;

        t[iter * numSamples + idx] = time;
        x[0 * numIters * numSamples + iter * numSamples + idx] = Sa;
        x[1 * numIters * numSamples + iter * numSamples + idx] = Sb;
        const int ib = static_cast<int>(time / bin_dt);         
        trajSaThreads[ib+idx*nbins] += Sa;
        trajSbThreads[ib+idx*nbins] += Sb;                      
        ++ntrajThreads[ib+idx*nbins];
    }

    // Termination markers.
    iters[idx]        = iter;
    isSampleDone[idx] = time >= endTime || isSampleDone[idx];
}


__device__ int countWarp(char isSampleDone) {

    unsigned bitCounts = __ballot_sync(0xFFFFFFFF, isSampleDone);

    return __popc(bitCounts);
}

// from Q1
template <class T>
__device__ T sumWarp(T a) {
    T sum = a;
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, 1);
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, 2);
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, 4);
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, 8);
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, 16);
    return sum;
}

template <class T>
__device__ T sumBlock(T a) {
    // TODO: 1.c) Compute the sum of values `a` for all threads within a block.
    //            Only threadIdx.x == 0 has to return the correct result.
    // NOTE: For 1.c) implement either this or `argMaxBlock`!

    // we are sure that there are 1024 threads all with meaningful data
    T result = sumWarp(a);

    __shared__ T sdata[32];

    if (threadIdx.x % 32 == 0)
        sdata[threadIdx.x / 32] = result;
    __syncthreads();

    if (threadIdx.x < 32) {
        result = sumWarp(sdata[threadIdx.x]);
    }

    return result;
}


template __device__  int sumWarp<int>(int a);
template __device__  double sumWarp<double>(double a);
template __device__  int sumBlock<int>(int a);
template __device__  double sumBlock<double>(double a);


/// Store the sum of the subarray isSampleDone[1024*b : 1024*b+1023] in blocksDoneCount[b].
__global__ void reduceIsDoneKernel(const char *isSampleDone, int *blocksDoneCount, int numSamples) {
    // TODO: Implement the reduction that computes how many samples in a block have completed.
    //       isSampleDone[sampleIdx] = 1 if sample has finished, 0 if not.
    //       blocksDoneCount[blockIdx] = 0..threads-1 (the value to compute).
    //       Feel free to reuse the code from Q1.
    //
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char done = idx < numSamples ? isSampleDone[idx] : 0;

    int count = countWarp(done);

    __shared__ int sdata[32];

    if (threadIdx.x % 32 == 0)
        sdata[threadIdx.x / 32] = count;
    __syncthreads();

    if (threadIdx.x < 32)
        count = sumWarp(sdata[threadIdx.x]);

    if (threadIdx.x == 0)
        blocksDoneCount[blockIdx.x] = count;
}


// TODO: Implement the binning mechanism.
//       
//       Add function prototypes to src/kernels.h, such that ssa.cu can access them.
// ...
// ...
// ...
// ...
// ...
// ...
// ...
// ...
// ...
// ...


__global__ void reduceTrajectoriesKernel(double* trajSaThreads, double* trajSbThreads, int* ntrajThreads,
                                         double* trajSaBlocks, double* trajSbBlocks, int* ntrajBlocks, int nbins) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // do this:
    // for(int t = 0; t < num_threads; ++t)
    // {
    //   for (int i = 0; i < nbins; ++i) {
    //     trajSaBlocks[i] += trajSaThreads[i+t*nbins];
    //     trajSbBlocks[i] += trajSbThreads[i+t*nbins];
    //     ntraj[i] += ntrajThreads[i+t*nbins];
    //   }
    // }

    for (int i = 0; i < nbins; ++i) {
        double Sa = idx < nbins ? trajSaThreads[i + idx*nbins] : 0.0;
        double Sb = idx < nbins ? trajSbThreads[i + idx*nbins] : 0.0;
        int n = idx < nbins ? ntrajThreads[i + idx*nbins] : 0;

        double sumSa = sumBlock(Sa);
        double sumSb = sumBlock(Sb);
        int sumN = sumBlock(n);

        if (threadIdx.x == 0 && blockIdx.x * blockDim.x < nbins) {
            trajSaBlocks[i + blockIdx.x*nbins] = sumSa;
            trajSbBlocks[i + blockIdx.x*nbins] = sumSb;
            ntrajBlocks[i + blockIdx.x*nbins] = sumN;
            // atomicAdd(trajSaBlocks + i, sumSa);
            // atomicAdd(trajSbBlocks + i, sumSb);
            // atomicAdd(ntrajBlocks + i, sumN);
        }

        // TODO: realize for higher dimensions...
        assert(gridDim.x < blockDim.x);

        if (blockIdx.x == 0) {
            Sa = idx * blockDim.x < nbins ? trajSaBlocks[i + idx*nbins] : 0.0;
            Sb = idx * blockDim.x < nbins ? trajSbBlocks[i + idx*nbins] : 0.0;
            n = idx * blockDim.x < nbins ? ntrajBlocks[i + idx*nbins] : 0;
            sumSa = sumBlock(Sa);
            sumSb = sumBlock(Sb);
            sumN = sumBlock(n);

            if (threadIdx.x == 0) {
                trajSaBlocks[i] = sumSa/sumN;
                trajSbBlocks[i] = sumSb/sumN;
                ntrajBlocks[i] = sumN;
            }
        }


    }


    

}