#include "ssa.h"
#include "kernels.h"
#include "utils.h"

#include <hiprand.h>

#include <algorithm>
#include <fstream>
#include <numeric>

static constexpr int NUM_SPECIES = 2;

void SSA_GPU::run()
{
    // Problem size.
    const int numIters = numItersPerPass;

    const int threads = 1024;  // Do not change this.
    const int blocks = (numSamples + threads - 1) / threads;
    if (blocks > 65536) {
        fprintf(stderr, "Number of samples larger than 64M not supported (block limit reached).\n");
        exit(1);
    }
    const long long memoryEstimate = 2ULL * numIters * threads * blocks * sizeof(float)
                                + 2*(numBins + numBins * (threads+1) * blocks) * sizeof(double)
                                + (numBins + numBins * (threads+1) * blocks) * sizeof(int);
    printf("SSA_GPU  numItersPerPass: %d  numSamples: %d  approx required memory: ~%.1fMB\n",
           numIters, numSamples, memoryEstimate / 1024. / 1024.);

    double* trajSaThreadsDev, *trajSaBlocksDev;
    double* trajSbThreadsDev, *trajSbBlocksDev;
    int* trajNThreadsDev, *trajNBlocksDev;
    float *uDev;            // Uniform random values vector.
    short *xDev;            // Species vector.
    float *tDev;            // Time vector.
    int *itersDev;          // Num iterations in simulation loop.
    char *isSampleDoneDev;  // isSampleDoneDev[sampleIdx] = 0 or 1.
    int *perBlockDoneDev;   // perBlockDoneDev[blockIdx] = number of samples done in the block blockIdx.
    int *perBlockDoneHost;  // A host copy.
    CUDA_CHECK(hipMalloc(&uDev, 2 * numIters * threads * blocks * sizeof(float)));
    CUDA_CHECK(hipMalloc(&uDev, 2 * numIters * threads * blocks * sizeof(float)));
    CUDA_CHECK(hipMalloc(&xDev, NUM_SPECIES * numSamples * numIters * sizeof(short)));
    CUDA_CHECK(hipMalloc(&tDev, numSamples * numIters * sizeof(float)));
    CUDA_CHECK(hipMalloc(&itersDev,      numSamples * sizeof(int)));
    CUDA_CHECK(hipMalloc(&isSampleDoneDev,      numSamples * sizeof(char)));
    CUDA_CHECK(hipMalloc(&perBlockDoneDev,      blocks * sizeof(int)));
    CUDA_CHECK(hipHostMalloc(&perBlockDoneHost, blocks * sizeof(int)));
    CUDA_CHECK(hipMalloc(&trajSaThreadsDev, numBins * threads * blocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&trajSbThreadsDev, numBins * threads * blocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&trajNThreadsDev, numBins * threads * blocks * sizeof(int)));
    CUDA_CHECK(hipMalloc(&trajSaBlocksDev, numBins * blocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&trajSbBlocksDev, numBins * blocks * sizeof(double)));
    CUDA_CHECK(hipMalloc(&trajNBlocksDev, numBins * blocks * sizeof(int)));

    CUDA_CHECK(hipMemset(itersDev,        0, numSamples * sizeof(int)));
    CUDA_CHECK(hipMemset(isSampleDoneDev, 0, numSamples * sizeof(char)));
    CUDA_CHECK(hipMemset(trajSaBlocksDev, 0.0, numBins * blocks * sizeof(double)));
    CUDA_CHECK(hipMemset(trajSbBlocksDev, 0.0, numBins * blocks * sizeof(double)));
    CUDA_CHECK(hipMemset(trajNBlocksDev, 0, numBins * blocks * sizeof(int)));

    hiprandGenerator_t generator;

    // Setup RNG.
    CURAND_CHECK(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_XORWOW));
    CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(generator, seed));

    printf("===== DIMERIZATION =====\n");
    const short Sa = 4 * omega;
    const short Sb = 0;

    // Set initial values of Sa, Sb and sample time (t=0 initially).
    initializationKernel<<<blocks, threads>>>(Sa, Sb, xDev, tDev, numSamples, numIters);

    // Evaluate samples in passes of `numIters` iterations.
    // (We cannot predict the total number of iterations that a sample might
    // need, so we allocate in advance buffers sufficient for `numIters` iterations.)
    for (int pass = 0; pass < 1000; ++pass) {
        // Generate random numbers needed by all threads for `numIters` iterations.
        CURAND_CHECK(hiprandGenerateUniform(generator, uDev, 2 * numIters * threads * blocks));

        // Evaluate up to `numIters` iterations.
        dimerizationKernel<<<blocks, threads>>>(
                pass, uDev, xDev, tDev, itersDev, isSampleDoneDev,
                endTime, omega, numIters, numSamples,
                trajSaThreadsDev,  trajSbThreadsDev,  trajNThreadsDev, numBins, dtBin);

        // TODO: Implement the binning mechanism.
        //       Use the sample trajectories xDev (which store Sa and Sb), tDev
        //       (trajectory time instances), itersDev (number of iterations
        //       for each sample, in this pass).
        //
        //       Allocate whatever memory you need, and implement the binning kernel (in kernels.cu) however you like.
        //       Make sure that the result is correct:
        //              a) data from all subtrajectories from all passes should be used,
        //              b) there must be no race condition when aggregating results.
        //       You can compare your results with HW4 solutions. See README
        //       for instructions on visualization.
        //
        //       The final result has to be stored in trajSa, trajSb and
        //       trajNumSteps (average Sa, average Sb, total number of samples,
        //       respectively).
        //
        //       Regarding the performance, take advantage of the GPU
        //       parallelism (i.e. do have multiple threads and multiple
        //       blocks).
        //

        // Check how many samples have finished.
        reduceIsDoneKernel<<<blocks, threads>>>(isSampleDoneDev, perBlockDoneDev, numSamples);
        CUDA_CHECK(hipMemcpy(perBlockDoneHost, perBlockDoneDev, blocks * sizeof(int), hipMemcpyDeviceToHost));
        int remaining = numSamples - std::accumulate(perBlockDoneHost, perBlockDoneHost + blocks, (int)0);
        printf("Execution Loop %d. Remaining samples: %d/%d\n",
               pass, remaining, numSamples);

        if (remaining == 0)
            break;
    }

    reduceTrajectoriesKernel<<<blocks, threads>>>(trajSaThreadsDev,  trajSbThreadsDev,  trajNThreadsDev,
        trajSaBlocksDev,  trajSbBlocksDev,  trajNBlocksDev, numBins);


    CUDA_CHECK(hipMemcpy(trajSa.data(), trajSaBlocksDev, numBins * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(trajSb.data(), trajSbBlocksDev, numBins * sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(trajNumSteps.data(), trajNBlocksDev, numBins * sizeof(int), hipMemcpyDeviceToHost));

    // TODO: Averaging. Store the result in trajSa, trajSb and trajNumSteps.
    //          trajSa[k] = average Sa in the time bin k
    //          trajSb[k] = average Sb in the time bin k
    //          trajNumSteps[k] = number of steps (Sa, Sb, t) in the time bin k

    CURAND_CHECK(hiprandDestroyGenerator(generator));

    // TODO: Deallocate all extra buffers you allocated.

    CUDA_CHECK(hipFree(trajSaThreadsDev));
    CUDA_CHECK(hipFree(trajSbThreadsDev));
    CUDA_CHECK(hipFree(trajNThreadsDev));
    CUDA_CHECK(hipFree(trajSaBlocksDev));
    CUDA_CHECK(hipFree(trajSbBlocksDev));
    CUDA_CHECK(hipFree(trajNBlocksDev));
    CUDA_CHECK(hipHostFree(perBlockDoneHost));
    CUDA_CHECK(hipFree(perBlockDoneDev));
    CUDA_CHECK(hipFree(isSampleDoneDev));
    CUDA_CHECK(hipFree(itersDev));
    CUDA_CHECK(hipFree(tDev));
    CUDA_CHECK(hipFree(xDev));
    CUDA_CHECK(hipFree(uDev));
}

void SSA_GPU::dumpTrajectoryToFile(const char *filename) {
    std::ofstream outfile(filename);

    int totalevals = 0;
    for (int i = 0; i < (int)trajSa.size(); ++i) {
        // Must rescale wrt omega.
        outfile << i*dtBin+dtBin/2 << ' '
                << (trajSa[i] / omega) << ' '
                << (trajSb[i] / omega) << '\n';
        totalevals += trajNumSteps[i];
    }
    printf("Average number of time steps per sample: %f\n", double(totalevals) / numSamples);
}
