#include <hip/hip_runtime.h>

__global__ void computeForcesKernel(int N, const double3 *p, double3 *f) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    // TODO: Copy the code from `nbody_a.cu` and fix the reduntant arithmetic operations.
    double3 tmp{0.0, 0.0, 0.0};
    for (int i = 0; i < N; ++i) {
        double dx = p[i].x - p[idx].x;
        double dy = p[i].y - p[idx].y;
        double dz = p[i].z - p[idx].z;
        // Instead of skipping the i == idx case, add 1e-150 to avoid division
        // by zero. (dx * inv_r will be exactly 0.0)
        double r = sqrt(1e-150 + dx * dx + dy * dy + dz * dz);
        double inv_r_3 = 1 / (r*r*r);
        tmp.x += dx * inv_r_3;
        tmp.y += dy * inv_r_3;
        tmp.z += dz * inv_r_3;
    }
    f[idx] = tmp;
}

void computeForces(int N, const double3 *p, double3 *f) {
    constexpr int numThreads = 1024;
    int numBlocks = (N + numThreads - 1) / numThreads;
    computeForcesKernel<<<numBlocks, numThreads>>>(N, p, f);
}
